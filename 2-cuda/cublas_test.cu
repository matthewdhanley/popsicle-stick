#include "hip/hip_runtime.h"
#include <hipblas.h>
#include "cuda_helper.h"

typedef float ftype;
extern "C"
void run(ftype *i1, ftype  *i2, ftype *o1, int d);


__global__ void p(ftype *i1, ftype *i2, ftype *o1, int d){

    hipblasHandle_t h;
    hipblasCreate(&h);
    ftype alpha = 1.0;
    ftype beta = 0.0;
    hipblasSgemm(h, HIPBLAS_OP_N, HIPBLAS_OP_N, d, d, d, &alpha, i1, d, i2, d, &beta, o1, d);
}

void run(ftype *i1, ftype  *i2, ftype *o1, int d){

    ftype *d_i1, *d_i2, *d_o1;
    int ds = d*d*sizeof(ftype);
    hipMalloc(&d_i1, ds);
    hipMalloc(&d_i2, ds);
    hipMalloc(&d_o1, ds);
    hipMemcpy(d_i1, i1, ds, hipMemcpyHostToDevice);
    hipMemcpy(d_i2, i2, ds, hipMemcpyHostToDevice);
    p<<<1,1>>>(d_i1, d_i2, d_o1, d);
    hipMemcpy(o1, d_o1, ds, hipMemcpyDeviceToHost);
    hipFree(d_i1);
    hipFree(d_i2);
    hipFree(d_o1);
}