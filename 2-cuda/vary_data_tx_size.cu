#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "gpu_err_check.h"
#include <float.h>

const unsigned long int MAX_OPS = 10000000000;
const int threads_per_block = 512;
const int n_blocks = 1000;

__global__ void strictly_ops(unsigned long int n_ops) {
    unsigned long int num_iterations = n_ops / threads_per_block / n_blocks / 2;
    float res;
    for (unsigned long int i = 0; i < num_iterations; i++) {
        res = 3.0 * 4.0 + 6.0;
    }
}


void cpu_strictly_ops(unsigned long int n_ops){
    float res;
    for (unsigned long int i = 0; i < n_ops/2; i++){
        res = 3.0 * 4.0 + 6.0;
    }
}


int main(){
    hipFree(0); // avoid spoofing profiler.
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    printf("n_ops, s_size, gpu, cpu\n");
    float milliseconds;
    for (unsigned long int i = 1; i <MAX_OPS; i*=2 ) {
        for (int j=2000; j<2000000; j*=10) {
            for (int k=0; k<10; k++) {
                printf("%lu,", i);
                printf("%d,",j);
                float* data = (float*) malloc(j);
                float* d_data;
                hipEventRecord(start);
                hipMalloc((void**) &d_data, j);
                hipMemcpy(d_data, data, j, hipMemcpyHostToDevice);
                strictly_ops << < n_blocks, threads_per_block >> > (i);
                hipDeviceSynchronize();
                hipMemcpy(data, d_data, j, hipMemcpyDeviceToHost);
                hipEventRecord(stop);
                hipEventSynchronize(stop);
                milliseconds = 0;
                hipEventElapsedTime(&milliseconds, start, stop);
                printf("%.7g,", milliseconds / 1000.0);

                hipEventRecord(start);
                cpu_strictly_ops(i);
                hipEventRecord(stop);
                hipEventSynchronize(stop);
                milliseconds = 0;
                hipEventElapsedTime(&milliseconds, start, stop);
                printf("%.7g\n", milliseconds / 1000.0);
            }
        }
    }

    return 0;
}