#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "gpu_err_check.h"
#include <float.h>

const unsigned long int MAX_OPS = 10000000000;
const int threads_per_block = 32;
const int n_blocks = 1000;

__global__ void naive(float* a, float* res) {
    int starting_point = blockIdx.x * blockDim.x;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    for (int i = 0; i < threads_per_block; i++) {
        res[tid] += a[starting_point + i];
    }
}


__global__ void coalesced(float* a, float* res, int offset) {
    int tid = (blockDim.x * blockIdx.x + threadIdx.x + offset) % threads_per_block;
    int starting_point = blockIdx.x * blockDim.x;
    for (int i = 0; i < threads_per_block; i++) {
        res[tid] += a[starting_point + i];
    }
}


int main(){
    hipFree(0); // avoid spoofing profiler.
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float* a = (float*) malloc(n_blocks * threads_per_block * sizeof(float));

    for (int i = 0; i < n_blocks * threads_per_block; i++){
        a[i] = 3.0;
    }

    float* d_a;
    float* d_res;


    printf("offset, naive, coalesced\n");
    for (int i=0; i<32; i++) {
        float milliseconds;
        for (int k = 0; k < 100; k++) {
            printf("%d,",i);
            hipMalloc((void **) &d_a, n_blocks * threads_per_block * sizeof(float));
            hipMalloc((void **) &d_res, n_blocks * threads_per_block * sizeof(float));
            hipMemcpy(d_a, a, n_blocks * threads_per_block * sizeof(float), hipMemcpyHostToDevice);

            hipEventRecord(start);
            naive <<< n_blocks, threads_per_block >>> (d_a, d_res);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipDeviceSynchronize();
            milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            printf("%.7g,", milliseconds / 1000.0);

            hipEventRecord(start);
            coalesced<<< n_blocks, threads_per_block >>> (d_a, d_res, i);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipDeviceSynchronize();
            milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            printf("%.7g\n", milliseconds / 1000.0);


            hipFree(&d_a);
        }
    }

    return 0;
}